// cuda_2d_fft.cu -  2D FFT Test for CUDA 
//
// Fred J. Frigo
// Marquette University
// 29-Jan-2025 
//
// See https://stackoverflow.com/questions/78967540/cuda-image-upsampling-with-fft-method
//
// To compile: nvcc -o cuda_2d_fft cuda_2d_fft.cu -lcufft 


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>   
#include <hipfft/hipfft.h>
#include <math.h>

typedef struct {
        float real;
        float imag;
} fcomplex;

int main(int argc, char **argv){

   int w = 1024, h = 1024;  // 2D FFT size is: (w x h )
   struct timeval t1, t2;
   double elapsedTime;
   fcomplex *data;

   //  initialize input data 
   data = (fcomplex *) malloc(sizeof(fcomplex) * w * h);
   int i, j;
   double pdata=0;
   for (i = 0; i < w; ++i){
     for (j = 0; j < h; ++j){
       data[i*w + j].real=(float)i;
       data[i*w + j].imag=(float)0;
       pdata+=data[i*w + j].real*data[i*w + j].real+data[i*w + j].imag*data[i*w + j].imag;
     }
   }
   // Show power of input signal to prove Parsevals Theorem
   printf("power of  input data is %f\n", pdata);

   // Input data buffer for device
   hipfftComplex *d_i;
   hipMalloc(&d_i, w*h*sizeof(hipfftComplex));

   // copy input data to device 
   hipMemcpy(d_i, data, w*h*sizeof(hipfftComplex), hipMemcpyHostToDevice);

   // create plan for 2D FFT
   hipfftHandle hf;
   hipfftResult r = hipfftPlan2d(&hf, h, w, HIPFFT_C2C); 
   
   // start timer 
   gettimeofday(&t1, NULL);

   // Perform Fourier Transform
   r = hipfftExecC2C(hf, d_i, d_i, HIPFFT_FORWARD);

   // stop timer 
   gettimeofday(&t2, NULL);

   // copy results from device to host 
   hipMemcpy(data, d_i, w*h*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

   // normalize results and calculate power of transform */
   double normalization=sqrt((double)w*h);
   double ptransform = 0;
   for (i = 0; i < w; ++i){
     for (j = 0; j < h; ++j){
       data[i*w+j].real/=normalization;
       data[i*w+j].imag/=normalization;
       ptransform+=data[i*w + j].real*data[i*w + j].real+data[i*w + j].imag*data[i*w + j].imag;
     }
   }
   printf("power of output data is %f\n", ptransform);
  
   free(data); 
   hipFree(d_i);

   // print the FFT time in millisec 
   elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
   elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;   // us to ms
   printf("FFT Time: %f ms.\n", elapsedTime);

   return 0;
 }
