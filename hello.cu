// hello.cu 
// 
//  Fred J. Frigo
//  01-Sep-2020
//
//  See section B19.4:
//  https://docs.nvidia.com/cuda/archive/9.1/pdf/CUDA_C_Programming_Guide.pdf`
//
//  To compile:  nvcc hello.cu -o hello
//


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA(float f)
{
   printf("Hello thread %d, f=%f\n", threadIdx.x, f);
}

int main()
{ 
   helloCUDA<<<1, 5>>>(1.2345f);
   hipDeviceSynchronize();
   printf("Hello CUDA!\n"); 
   return 0;
}